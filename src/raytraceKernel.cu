#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

//#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
//#define  printf(f, ...) ((void)(f, __VA_ARGS__),0)  
//#endif

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
  ray r;
  r.origin = eye; 
  float sx, sy;
  //printf("%d,%d",x,y);
  sx = (float)x/((float)resolution.x-1);
  sy = (float)y/((float)resolution.y-1);
  glm::vec3 A = glm::normalize(glm::cross(view,up));
  glm::vec3 B = glm::normalize(glm::cross(A,view));
  double radian = (float)fov.y/180.0*PI;
  float tmp = tan(radian) * glm::length(view)/glm::length(B);
  glm::vec3 V = B;
  V*= tmp;
  tmp = tan(radian) * (float)resolution.x/(float)resolution.y*glm::length(view)/glm::length(A);
  glm::vec3 H = A;
  H*=tmp;
  H *= (2.0*sx-1);
  V *= (1-2.0*sy);
  glm::vec3 p = eye + view + H + V;
  r.direction = p-eye;
  r.direction = glm::normalize(r.direction);
  //r.direction = glm::normalize(r.direction);
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}
/////shadow check
__host__ __device__ bool ShadowRayUnblocked(glm::vec3 point,glm::vec3 lightPos,staticGeom* geoms, int numberOfGeoms,material* mats)
{
	//return true;
	float tmpDist = -1;
	glm::vec3 tmpnormal;
	glm::vec3 intersectionPoint;
	ray r; r.origin = point;
	r.direction = lightPos-point; r.direction = glm::normalize(r.direction);	
	float objDist = -1;
	for(int i = 0;i<numberOfGeoms;++i)
	{
		//if light source no need to check
		if(mats[i].emittance >0)
			continue;
		if(geoms[i].type == SPHERE )
		{			
			//if same location as light, no need to check
			if(glm::length(geoms[i].translation - lightPos)<0.001)
				continue;
			tmpDist = sphereIntersectionTest(geoms[i],r,intersectionPoint,tmpnormal);							
			if(abs(tmpDist+1)<EPSILON || abs(tmpDist)<=0.03)
				continue;// obj not in shadow
			else
			{
				objDist = tmpDist;
			}
			
		}
		else if(geoms[i].type == CUBE)
		{		
			if(glm::length(geoms[i].translation - lightPos)<0.0001)
				continue;
			tmpDist = boxIntersectionTest(geoms[i],r,intersectionPoint,tmpnormal);		
			if(abs(tmpDist+1)< EPSILON || abs(tmpDist)<=0.03)	
				continue;
			else
			{
				objDist = tmpDist;
			}				
		}
	}
	if(objDist == -1)
		return true;
	else
		return false;	
}

//recursive raytrace
__host__ __device__ void raytrace(ray Ri,glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3& color,
                            staticGeom* geoms, int numberOfGeoms,material* mats,int* lightIndex,int lightNum)
{
	/////////////variables//////////////	
	ray Rr; //reflect ray
	glm::vec3 intersectionPoint(0,0,0);
	glm::vec3 normal(0,0,0);
	glm::vec3 tmpnormal(0,0,0);
	float interPointDist = -1;
	float tmpDist = -1;
	glm::vec3 diffuseColor(0,0,0);
	glm::vec3 specularColor(0,0,0);  
	glm::vec3 reflectedColor(0,0,0);
	glm::vec3 refractColor(0,0,0);
	glm::vec3 localColor(0,0,0);
	int nearestObjIndex = -1; // nearest intersect object index
	glm::vec3 ambient(ambientColorR,ambientColorG,ambientColorB);
	ambient *= Kambient;
	int nearestLight = -1;
	float lightDist = -1;
	float tmplightDist = -1;
	////////////////////////////////////////////
	color = glm::vec3(0,0,0);

	for(int i = 0;i<numberOfGeoms;++i)
	{
		//if the object is light, continue to another
		if(geoms[i].type == SPHERE )
		{

			tmpDist =  sphereIntersectionTest(geoms[i],Ri,intersectionPoint,tmpnormal);
			if(tmpDist!=-1 &&(interPointDist==-1 ||(interPointDist!=-1 && tmpDist<interPointDist)))
			{
					
				if(mats[i].emittance>0)
				{
					nearestLight = i;
				}
				else
				{
					nearestLight = -1;
					interPointDist = tmpDist;
					normal = tmpnormal;
					nearestObjIndex = i;
				}
			}
				
		}
		else if(geoms[i].type == CUBE)
		{
			
			tmpDist = boxIntersectionTest(geoms[i],Ri,intersectionPoint,tmpnormal);
			if(tmpDist!=-1 &&(interPointDist==-1 ||(interPointDist!=-1 && tmpDist<interPointDist)))
			{
				if(mats[i].emittance>0)
					nearestLight = i;
				else
				{
					nearestLight = -1;
					interPointDist = tmpDist;
					normal = tmpnormal;
					nearestObjIndex = i;
				}
			}
				
		}
	}

	//if first ray didn't hit any object,color set to light / bg color
	if(interPointDist == -1 ||(interPointDist!=-1 && nearestLight!=-1))
	{
		if(nearestLight != -1)
		{
			color = mats[nearestLight].color;
		}
		else
			color = glm::vec3(bgColorR,bgColorG,bgColorB);
		return;
	}
	//did hit object
	else
	{			
		//printf("%d ",nearestObjIndex);
		/*colors[index] = glm::vec3(abs(normal.x),abs(normal.y),abs(normal.z));
		return;*/
		Rr.direction = normal; 
		Rr.direction *= glm::dot(Ri.direction,normal);
		Rr.direction *= -2.0;
		Rr.direction += Ri.direction;
		Rr.origin = intersectionPoint;	
		localColor += ambient* mats[nearestObjIndex].color;
			
		//shadow check
		for(int j = 0;j<lightNum;++j)
		{
			
			if(ShadowRayUnblocked(intersectionPoint,geoms[lightIndex[j]].translation,geoms,numberOfGeoms,mats) == true)
			{
				//not in shadow
				//add diffuse 			
				diffuseColor = mats[nearestObjIndex].color;								
				glm::vec3 L = glm::normalize(geoms[lightIndex[j]].translation - intersectionPoint);
				float diffuseCon = glm::dot(normal,L);
				if(diffuseCon<0)
					diffuseColor = glm::vec3(0,0,0);
				else
				{
					diffuseColor *= diffuseCon;
					diffuseColor *= Kdiffuse;
					diffuseColor *= mats[lightIndex[j]].color;
				}			
				localColor += diffuseColor;
			/*	specularColor *= pow(glm::dot(Rr.direction,(cam.position-intersectionPoint)),mats[nearestObjIndex].specularExponent);
				specularColor *= mats[lightIndex[j]].color;
				localColor += specularColor;*/
			}
		}
	}
	color += localColor;	
}



//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms,material* mats,int* lightIndex,int lightNum){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if((x<=resolution.x && y<=resolution.y)){
		ray Ri; //indice ray
		Ri = raycastFromCameraKernel(resolution, time, x,y,cam.position, cam.view, cam.up, cam.fov);
		raytrace(Ri,resolution,time,cam,rayDepth,colors[index],geoms,numberOfGeoms,mats,lightIndex,lightNum);
   }
}


//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  //material
  material* matList = new material[numberOfGeoms];
  int lightNum = 0;
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
	matList[i] = materials[newStaticGeom.materialid];
	if(matList[i].emittance >0)
		lightNum++;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  int* lightIndex = new int[lightNum];
  int lin = 0;
  for(int i = 0;i<numberOfGeoms;i++)
  {
	  if(matList[i].emittance ==0) continue;
	  lightIndex[lin] = i;
	  lin++;
  }
  int* cudalightindex = NULL;
  hipMalloc((void**)&cudalightindex,lightNum*sizeof(int));
  hipMemcpy(cudalightindex,lightIndex,lightNum*sizeof(int),hipMemcpyHostToDevice);
  material* cudamat = NULL;
  hipMalloc((void**)&cudamat,numberOfGeoms*sizeof(material));
  hipMemcpy(cudamat,matList,numberOfGeoms*sizeof(material),hipMemcpyHostToDevice);
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms,cudamat,cudalightindex,lightNum);

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  delete geomList;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}